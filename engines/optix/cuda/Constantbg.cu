#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>

#include "renderer/VolumeRenderer.h"
#include "CommonStructs.h"

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, bg_color, , );
rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Constant background
RT_PROGRAM void miss()
{
    // Volume
    const float4 volumeContribution = getVolumeContribution(ray, INFINITY);
    float4 result = make_float4(bg_color, 0.f);
    if(volumeContribution.w > 0.f)
        result = volumeContribution;
    prd_radiance.result = make_float3(result);
}

// Environment map background
RT_PROGRAM void envmap_miss()
{
    float4 result = getVolumeContribution(ray, INFINITY);
    if(result.w == 0.f)
    {
        float theta = atan2f( ray.direction.x, ray.direction.z );
        float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
        float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
        float v     = -0.5f * ( 1.0f + sin(phi) );
        result = tex2D(envmap, u, v);
    }
    prd_radiance.result = make_float3(result);
}
